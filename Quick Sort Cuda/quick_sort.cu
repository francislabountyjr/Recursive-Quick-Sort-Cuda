#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH 24
#define INSERTION_SORT 32

// use selection sort when data reaches the max depth level
__device__ void selection_sort(unsigned int* data, int left, int right)
{
	for (int i = left; i <= right; ++i)
	{
		unsigned min_val = data[i];
		int min_idx = i;

		// find the smallest value in the range [left, right]
		for (int j = i + 1; j <= right; ++j)
		{
			unsigned val_j = data[j];
			if (val_j < min_val)
			{
				min_idx = j;
				min_val = val_j;
			}
		}

		// swap the values
		if (i != min_idx)
		{
			data[min_idx] = data[i];
			data[i] = min_val;
		}
	}
}

// quicksort algorithm using dynamic parallelism sorting recursively until the max depth is reached
__global__ void cdp_simple_quicksort(unsigned int* data, int left, int right, int depth)
{
	if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT)
	{
		selection_sort(data, left, right);
		return;
	}

	unsigned int* lptr = data + left;
	unsigned int* rptr = data + right;
	unsigned int pivot = data[(left + right) / 2];

	// partitioning
	while (lptr <= rptr)
	{
		// find the next left and right hand values to swap
		unsigned int lval = *lptr;
		unsigned int rval = *rptr;

		// move the left pointer as long as the pointed element is less than the pivot
		while (lval < pivot)
		{
			lptr++;
			lval = *lptr;
		}

		// move the right pointer as long as the pointed element is larger than the pivot
		while (rval > pivot)
		{
			rptr--;
			rval = *rptr;
		}

		// if the points are valid, conduct the swap
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	// recursive set up
	int nright = rptr - data;
	int nleft = lptr - data;

	// launch a new block to sort the left part
	if (left < (rptr - data))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		cdp_simple_quicksort<<<1, 1, 0, s>>>(data, left, nright, depth + 1);
		hipStreamDestroy(s);
	}

	// launch a new block to sort the right part
	if ((lptr - data) < right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		cdp_simple_quicksort<<<1, 1, 0, s1>>>(data, nleft, right, depth + 1);
		hipStreamDestroy(s1);
	}
}

// call quicksort kernel from the host
void run_qsort(unsigned int* data, unsigned int nitems)
{
	// prepare CDP for the max depth 'MAX_DEPTH'
	checkCudaErrors(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

	// launch kernel on device
	int left = 0;
	int right = nitems - 1;
	printf("Launching kernel on the GPU\n");
	cdp_simple_quicksort<<<1, 1>>>(data, left, right, 0);
	checkCudaErrors(hipDeviceSynchronize());
}

// initialize data on host
void initialize_data(unsigned int* dst, unsigned int nitems, int seed)
{
	srand(seed);

	// fill dst with random values
	for (unsigned i = 0; i < nitems; i++)
	{
		dst[i] = rand() % nitems;
	}
}

// verify the results
void check_results(int n, unsigned int* results_d)
{
	unsigned int* results_h = new unsigned[n];
	checkCudaErrors(hipMemcpy(results_h, results_d, n * sizeof(unsigned), hipMemcpyDeviceToHost));

	for (int i = 1; i < n; i++)
	{
		if (results_h[i - 1] > results_h[i])
		{
			printf("Invalid item [%d]: %d greater than %d\n", i - 1, results_h[i - 1], results_h[i]);
			exit(EXIT_FAILURE);
		}
	}

	printf("OK\n");
	delete[] results_h;
}

int main()
{
	int num_items = 2048;

	// create input data
	unsigned int* h_data = 0;
	unsigned int* d_data = 0;

	// allocate CPU memory and initialize data
	h_data = (unsigned int*)malloc(num_items * sizeof(unsigned int));
	initialize_data(h_data, num_items, 2021);

	// allocate GPU memory
	checkCudaErrors(hipMalloc((void**)&d_data, num_items * sizeof(unsigned int)));
	checkCudaErrors(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

	// execute
	printf("Running quicksort on %d elements\n", num_items);
	run_qsort(d_data, num_items);

	// check the result
	printf("Validating results: ");
	check_results(num_items, d_data);

	// cleanup
	checkCudaErrors(hipFree(d_data));
	free(h_data);
}